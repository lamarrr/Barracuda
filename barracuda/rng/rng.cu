/**
 * @file rng.cu
 * @author Basit Ayantunde (rlamarrr@gmail.com)
 * @brief
 * @version 0.1.0
 * @date 2019-11-28
 *
 * @copyright Copyright (c) 2019
 *
 */

#include "barracuda/rng/rng.cuh"

namespace bcuda {
namespace pseudo_rng {

Uniform::Uniform() : generator_{nullptr} {
  BCUDA_ENSURE_TRUE_STR(
      hiprandCreateGenerator(&generator_, HIPRAND_RNG_PSEUDO_DEFAULT) ==
          hiprandStatus_t::HIPRAND_STATUS_SUCCESS,
      "Unable to create Pseudo Random Number Generator");
}

Uniform::~Uniform() noexcept {
  BCUDA_ENSURE_TRUE_STR(hiprandDestroyGenerator(generator_) ==
                            hiprandStatus_t::HIPRAND_STATUS_SUCCESS,
                        "Error destroying pseudo-random number generator");
  generator_ = nullptr;
}

void Uniform::Seed(size_t seed) {
  BCUDA_ENSURE_TRUE_STR(hiprandSetPseudoRandomGeneratorSeed(generator_, seed) ==
                            hiprandStatus_t::HIPRAND_STATUS_SUCCESS,
                        "Unable to set pseudo-random number generator seed");
}

void Uniform::GenerateSequence_(float *data, size_t n) {
  BCUDA_ENSURE_TRUE_STR(hiprandGenerateUniform(generator_, data, n) ==
                            hiprandStatus_t::HIPRAND_STATUS_SUCCESS,
                        "Error generating Pseudo-random sequence for tensor");
}

void Uniform::GenerateSequence_(double *data, size_t n) {
  BCUDA_ENSURE_TRUE_STR(hiprandGenerateUniformDouble(generator_, data, n) ==
                            hiprandStatus_t::HIPRAND_STATUS_SUCCESS,
                        "Error generating Pseudo-random sequence for tensor");
}

}  // namespace  pseudo_rng

}  // namespace bcuda
